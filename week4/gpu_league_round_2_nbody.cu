#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 2: n-body simulation
//////////////////////////////////////////////////////////////////////////
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="Slim_Shaders";
	std::string author_1="Andrw_Yang";
	std::string author_2="Matthew_Kenney";
};

//////////////////////////////////////////////////////////////////////////
////Here is a sample function implemented on CPU for n-body simulation.

__host__ void N_Body_Simulation_CPU_Poorman(double* pos_x,double* pos_y,double* pos_z,		////position array
											double* vel_x,double* vel_y,double* vel_z,		////velocity array
											double* acl_x,double* acl_y,double* acl_z,		////acceleration array
											const double* mass,								////mass array
											const int n,									////number of particles
											const double dt,								////timestep
											const double epsilon_squared)					////epsilon to avoid 0-denominator
{		
	////Step 1: set particle accelerations to be zero
	memset(acl_x,0x00,sizeof(double)*n);
	memset(acl_y,0x00,sizeof(double)*n);
	memset(acl_z,0x00,sizeof(double)*n);

	////Step 2: traverse all particle pairs and accumulate gravitational forces for each particle from pairwise interactions
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			////skip calculating force for itself
			if(i==j) continue;

			////r_ij=x_j-x_i
			double rx=pos_x[j]-pos_x[i];
			double ry=pos_y[j]-pos_y[i];
			double rz=pos_z[j]-pos_z[i];

			////a_ij=m_j*r_ij/(r+epsilon)^3, 
			////noticing that we ignore the gravitational coefficient (assuming G=1)
			double dis_squared=rx*rx+ry*ry+rz*rz;
			double dis_test = dis_squared+epsilon_squared; 
			double dis_6 = dis_test * dis_test * dis_test; 
			double one_over_dis_cube=rsqrt(dis_6); 
			double ax=mass[j]*rx*one_over_dis_cube;
			double ay=mass[j]*ry*one_over_dis_cube;
			double az=mass[j]*rz*one_over_dis_cube;

			////accumulate the force to the particle
			acl_x[i]+=ax;
			acl_y[i]+=ay;
			acl_z[i]+=az;
		}
	}

	////Step 3: explicit time integration to update the velocity and position of each particle
	for(int i=0;i<n;i++){
		////v_{t+1}=v_{t}+a_{t}*dt
		vel_x[i]+=acl_x[i]*dt;
		vel_y[i]+=acl_y[i]*dt;
		vel_z[i]+=acl_z[i]*dt;

		////x_{t+1}=x_{t}+v_{t}*dt
		pos_x[i]+=vel_x[i]*dt;
		pos_y[i]+=vel_y[i]*dt;
		pos_z[i]+=vel_z[i]*dt;
	}
}


//////////////////////////////////////////////////////////////////////////
////TODO 1: your GPU variables and functions start here

// Compute Acceleration from Force interaction between two bodies  
__device__ double3 findAccel(const double4 ipos, const double4 jpos, //// Body comparing to  
						  const double epsilon_squared, double3 ai)
{
	// ipos -> position (and mass) of body i
	// jpos -> position (and mass) of body j
	// epsilon_squared -> softening factor
	// ai -> acceleration of body i to update

	// Compute the Denominator of the Acceleration Update
	double rx = jpos.x - ipos.x;
	double ry = jpos.y - ipos.y;
	double rz = jpos.z - ipos.z;
	double r2 = rx * rx + ry * ry + rz * rz + epsilon_squared;
	double r_6 = r2 * r2 * r2; 
	double directionless_ai = jpos.w * rsqrt(r_6); 

	// Compute the change in acceleration:
	ai.x += rx * directionless_ai;
	ai.y += ry * directionless_ai;
	ai.z += rz * directionless_ai;

	return ai;

}


// Computes Velocity given Acceleration for a single body 
// Given time step, velocity, and acceleration for that body 
__device__ double3 findV(double3 vel, double3 acc, const double dt)
{
	// update velocity 
	vel.x += acc.x * dt;
	vel.y += acc.y * dt;
	vel.z += acc.z * dt;

	return vel;

}

// Computes the acceleration changes to all bodies for a given time step.
// Implements a tiling approach in order to achieve shared memory speedups.
__global__ void tileForceBodies(double4* pos, double3 *vel, double3 *acc, 
								const double epsilon_squared, 
								const double dt, 
								const int particle_n) 
{
	int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	int local_tid = threadIdx.x;

	if (global_tid < particle_n) {

		// This thread's information:
		double4 this_pos = pos[global_tid]; // w (mass), x, y, z
		double3 this_vel = vel[global_tid]; // current body's velocity
		double3 this_acc; // current acceleration (set via N-body computation)
		this_acc.x = 0.0;
		this_acc.y = 0.0;
		this_acc.z = 0.0;


		// 1 x blockDim shared memory
		extern __shared__ double4 bodyData[];
	
		// Load shared memory 
		#pragma unroll 4
		for(int i = 0; i < particle_n; i+=blockDim.x) { // divides particles into N/blockDim chunks

			// load position values for blockDim particles into shared memory:
			bodyData[local_tid] = pos[i + local_tid]; // move blockDim slots ahead on each outer loop execution
			__syncthreads();

			// Calculate interactions between current body & all bodies j in the domain j ∈ [i, i + blockDim) 
			#pragma unroll 32
			for(int j = 0; j < blockDim.x; j++) {
				double4 jpos = bodyData[j];
				this_acc = findAccel(this_pos, jpos, epsilon_squared, this_acc);
			}
			__syncthreads();
		}

		// Find velocity 
		this_vel = findV(this_vel, this_acc, dt);

		// write back to global memory:
		acc[global_tid] = this_acc;
		vel[global_tid] = this_vel;
	}
}

// Kernel Function to update the positions of all bodies once acceleration update has finished
__global__ void updatePositions(double4* pos, double3* vel, const double dt) {

	int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	pos[global_tid].x += vel[global_tid].x * dt;
	pos[global_tid].y += vel[global_tid].y * dt;
	pos[global_tid].z += vel[global_tid].z * dt;

}



////Your implementations end here
//////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////
////Test function for n-body simulator
ofstream out;

//////////////////////////////////////////////////////////////////////////
////Please do not change the values below
const double dt=0.001;							////time step
const int time_step_num=10;						////number of time steps
const double epsilon=1e-2;						////epsilon added in the denominator to avoid 0-division when calculating the gravitational force
const double epsilon_squared=epsilon*epsilon;	////epsilon squared

////We use grid_size=4 to help you debug your code, change it to a bigger number (e.g., 16, 32, etc.) to test the performance of your GPU code
const unsigned int grid_size=20;					////assuming particles are initialized on a background grid
const unsigned int particle_n=pow(grid_size,3);	////assuming each grid cell has one particle at the beginning

// Thread Count is min of particle_n and 32 (so as not to spawn excess threads in the case of a small number of bodies)
const unsigned int thread_count = min(particle_n, 32);

__host__ void Test_N_Body_Simulation()
{
	////initialize position, velocity, acceleration, and mass
	//printf("Using %d threads per block\n", thread_count);
	//printf("Using %d blocks\n\n", (int)ceil(double(particle_n)/double(thread_count)));
	
	double* pos_x=new double[particle_n];
	double* pos_y=new double[particle_n];
	double* pos_z=new double[particle_n];
	////initialize particle positions as the cell centers on a background grid
	double dx=1.0/(double)grid_size;
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_x[index]=dx*(double)i;
				pos_y[index]=dx*(double)j;
				pos_z[index]=dx*(double)k;
			}
		}
	}

	double* vel_x=new double[particle_n];
	memset(vel_x,0x00,particle_n*sizeof(double));
	double* vel_y=new double[particle_n];
	memset(vel_y,0x00,particle_n*sizeof(double));
	double* vel_z=new double[particle_n];
	memset(vel_z,0x00,particle_n*sizeof(double));

	double* acl_x=new double[particle_n];
	memset(acl_x,0x00,particle_n*sizeof(double));
	double* acl_y=new double[particle_n];
	memset(acl_y,0x00,particle_n*sizeof(double));
	double* acl_z=new double[particle_n];
	memset(acl_z,0x00,particle_n*sizeof(double));

	double* mass=new double[particle_n];
	for(int i=0;i<particle_n;i++) {
		mass[i] = 100.0;
	}


	//////////////////////////////////////////////////////////////////////////
	////Default implementation: n-body simulation on CPU
	////Comment the CPU implementation out when you test large-scale examples
	auto cpu_start=chrono::system_clock::now();
	cout<<"Total number of particles: "<<particle_n<<endl;
	cout<<"Tracking the motion of particle "<<particle_n/2<<endl;

	for(int i=0;i<time_step_num;i++){

		N_Body_Simulation_CPU_Poorman(pos_x,pos_y,pos_z,vel_x,vel_y,vel_z,acl_x,acl_y,acl_z,mass,particle_n,dt,epsilon_squared);
		cout<<"pos on timestep "<<i<<": "<<pos_x[particle_n/2]<<", "<<pos_y[particle_n/2]<<", "<<pos_z[particle_n/2]<<endl;

	}

	auto cpu_end=chrono::system_clock::now();
	chrono::duration<double> cpu_time=cpu_end-cpu_start;
	cout<<"CPU runtime: "<<cpu_time.count()*1000.<<" ms."<<endl;

	//////////////////////////////////////////////////////////////////////////
	// Creating double values like CPU and moving to GPU 

	double4* pos_host = new double4[particle_n]; 
	double3* vel_host= new double3[particle_n]; 
	double3* acl_host = new double3[particle_n];

	// Set position and mass data in pos_gpu 
	for(unsigned int k=0;k<grid_size;k++){
		for(unsigned int j=0;j<grid_size;j++){
			for(unsigned int i=0;i<grid_size;i++){
				unsigned int index=k*grid_size*grid_size+j*grid_size+i;
				pos_host[index].x = dx*(double)i; 
				pos_host[index].y = dx*(double)j; 
				pos_host[index].z = dx*(double)k; 
			}
		}
	}

	for(int i=0;i<particle_n;i++) {
		pos_host[i].w=100.0;
	}
	
	// set velocity and acceleration vectors to 0 
	for(int i=0; i<particle_n; i++){
		vel_host[i].x = 0;
		vel_host[i].y = 0;
		vel_host[i].z = 0;
	}

	// Copy vectors over to GPU  
	double4* pos_gpu; 
	double3* vel_gpu; 
	double3* acl_gpu; 

	hipMalloc((void**)&pos_gpu, particle_n * sizeof(double4)); 
	hipMalloc((void**)&vel_gpu, particle_n * sizeof(double3)); 
	hipMalloc((void**)&acl_gpu, particle_n * sizeof(double3)); 

	hipMemcpy(pos_gpu, pos_host, particle_n*sizeof(double4), hipMemcpyHostToDevice);
	hipMemcpy(vel_gpu, vel_host, particle_n*sizeof(double3), hipMemcpyHostToDevice); 

	//////////////////////////////////////////////////////////////////////////
	////Your implementation: n-body simulator on GPU
	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//////////////////////////////////////////////////////////////////////////
	////TODO 2: Your GPU functions are called here
	////Requirement: You need to copy data from the CPU arrays, conduct computations on the GPU, and copy the values back from GPU to CPU
	////The final positions should be stored in the same place as the CPU n-body function, i.e., pos_x, pos_y, pos_z
	////The correctness of your simulation will be evaluated by comparing the results (positions) with the results calculated by the default CPU implementations
	//////////////////////////////////////////////////////////////////////////
	int num_blocks = ceil((double)particle_n/(double)thread_count);
	
	cout<<"\nTotal number of particles: "<<particle_n<<endl;
	cout<<"Tracking the motion of particle "<<particle_n/2<<endl;
	cout<<"Print statements disabled "<<endl;

	// Step through time 
	for(int i=0;i<time_step_num;i++){

		// Here, we synchronize global memory before updating positions to avoid
		// Read-after-write conflicts for large values of particle_n
		tileForceBodies<<<num_blocks, thread_count, thread_count * sizeof(double4)>>>
			(pos_gpu, vel_gpu, acl_gpu, epsilon_squared, dt, particle_n);
	
		// Synchronize and write to global memory
		hipDeviceSynchronize();
		updatePositions<<<num_blocks, thread_count>>>(pos_gpu, vel_gpu, dt);
		hipDeviceSynchronize();
		
		// Print Results to console (comment out to test performance)
		hipMemcpy(pos_host, pos_gpu, particle_n*sizeof(double4), hipMemcpyDeviceToHost);
		cout<<"pos on timestep "<<i<<": "<<pos_host[particle_n/2].x<<", "<<pos_host[particle_n/2].y<<", "<<pos_host[particle_n/2].z<<endl;
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);

	//////////////////////////////////////////////////////////////////////////
	// One Memcpy at the end of all kernel calls to return data to the host:
	hipMemcpy(pos_host, pos_gpu, particle_n*sizeof(double4), hipMemcpyDeviceToHost);

	// NOTE: Since we used our own double4 to store the values of our particles, we altered the write statement here
	// to reflect the way that we stored our values.
	out<<"R0: "<<pos_host[particle_n/2].x<<" " <<pos_host[particle_n/2].y<<" " <<pos_host[particle_n/2].z<<endl;
	out<<"T1: "<<gpu_time<<endl;
}

int main()
{

	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_2_nbody.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Test_N_Body_Simulation();

	return 0;
}
