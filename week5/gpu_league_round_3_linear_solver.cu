//////////////////////////////////////////////////////////////////////////
////This is the code implementation for GPU Premier League Round 3: sparse linear solver
//////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="";
	std::string author_1="Name_1";
	std::string author_2="Name_2";
	std::string author_3="Name_3";	////optional
};

//////////////////////////////////////////////////////////////////////////
////TODO: Read the following three CPU implementations for Jacobi, Gauss-Seidel, and Red-Black Gauss-Seidel carefully
////and understand the steps for these numerical algorithms
//////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////
////These are the global variables that define the domain of the problem to solver
////You will need to use these parameters or macros in your GPU implementations
//////////////////////////////////////////////////////////////////////////

const int n=8;							////grid size, we will change this value to up to 256 to test your code
const int g=1;							////padding size
const int s=(n+2*g)*(n+2*g);			////array size
#define I(i,j) (i+g)*(n+2*g)+(j+g)		////2D coordinate -> array index
#define B(i,j) i<0||i>=n||j<0||j>=n		////check boundary
const bool verbose=true;				////set false to turn off print for x and residual
const double tolerance=1e-3;			////tolerance for the iterative solver

//////////////////////////////////////////////////////////////////////////
////The following are three sample implementations for CPU iterative solvers
void Jacobi_Solver(double* x,const double* b)
{
	double* buf=new double[s];
	memcpy(buf,x,sizeof(double)*s);
	double* xr=x;			////read buffer pointer
	double* xw=buf;			////write buffer pointer
	int iter_num=0;			////iteration number
	int max_num=1e5;		////max iteration number
	double residual=0.0;	////residual

	do{
		////update x values using the Jacobi iterative scheme
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				xw[I(i,j)]=(b[I(i,j)]+xr[I(i-1,j)]+xr[I(i+1,j)]+xr[I(i,j-1)]+xr[I(i,j+1)])/4.0;
			}
		}

		////calculate residual
		residual=0.0;
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				residual+=pow(4.0*xw[I(i,j)]-xw[I(i-1,j)]-xw[I(i+1,j)]-xw[I(i,j-1)]-xw[I(i,j+1)]-b[I(i,j)],2);
			}
		}

		if(verbose)cout<<"res: "<<residual<<endl;

		////swap the buffers
		double* swap=xr;
		xr=xw;
		xw=swap;
		iter_num++;
	}while(residual>tolerance&&iter_num<max_num);	

	x=xr;

	cout<<"Jacobi solver converges in "<<iter_num<<" iterations, with residual "<<residual<<endl;

	delete [] buf;
}

void Gauss_Seidel_Solver(double* x,const double* b)
{
	int iter_num=0;			////iteration number
	int max_num=1e5;		////max iteration number
	double residual=0.0;	////residual

	do{
		////update x values using the Gauss-Seidel iterative scheme
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				x[I(i,j)]=(b[I(i,j)]+x[I(i-1,j)]+x[I(i+1,j)]+x[I(i,j-1)]+x[I(i,j+1)])/4.0;
			}
		}

		////calculate residual
		residual=0.0;
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				residual+=pow(4.0*x[I(i,j)]-x[I(i-1,j)]-x[I(i+1,j)]-x[I(i,j-1)]-x[I(i,j+1)]-b[I(i,j)],2);
			}
		}

		if(verbose)cout<<"res: "<<residual<<endl;
		iter_num++;
	}while(residual>tolerance&&iter_num<max_num);	

	cout<<"Gauss-Seidel solver converges in "<<iter_num<<" iterations, with residual "<<residual<<endl;
}

void Red_Black_Gauss_Seidel_Solver(double* x,const double* b)
{
	int iter_num=0;			////iteration number
	int max_num=1e5;		////max iteration number
	double residual=0.0;	////residual

	do{
		////red G-S
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				if((i+j)%2==0)		////Look at this line!
					x[I(i,j)]=(b[I(i,j)]+x[I(i-1,j)]+x[I(i+1,j)]+x[I(i,j-1)]+x[I(i,j+1)])/4.0;
			}
		}

		////black G-S
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				if((i+j)%2==1)		////And this line!
					x[I(i,j)]=(b[I(i,j)]+x[I(i-1,j)]+x[I(i+1,j)]+x[I(i,j-1)]+x[I(i,j+1)])/4.0;
			}
		}

		////calculate residual
		residual=0.0;
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				residual+=pow(4.0*x[I(i,j)]-x[I(i-1,j)]-x[I(i+1,j)]-x[I(i,j-1)]-x[I(i,j+1)]-b[I(i,j)],2);
			}
		}

		if(verbose)cout<<"res: "<<residual<<endl;
		iter_num++;
	}while(residual>tolerance&&iter_num<max_num);	

	cout<<"Red-Black Gauss-Seidel solver converges in "<<iter_num<<" iterations, with residual "<<residual<<endl;
}

//////////////////////////////////////////////////////////////////////////
////In this function, we are solving a Poisson equation -laplace(p)=b, with p=x^2+y^2 and b=4.
////The boundary conditions are set on the one-ring ghost cells of the grid
//////////////////////////////////////////////////////////////////////////

void Test_CPU_Solvers()
{
	double* x=new double[s];
	memset(x,0x0000,sizeof(double)*s);
	double* b=new double[s];
	for(int i=-1;i<=n;i++){
		for(int j=-1;j<=n;j++){
			b[I(i,j)]=4.0;		////set the values for the right-hand side
		}
	}

	//////////////////////////////////////////////////////////////////////////
	////test Jacobi
	for(int i=-1;i<=n;i++){
		for(int j=-1;j<=n;j++){
			if(B(i,j))
				x[I(i,j)]=(double)(i*i+j*j);	////set boundary condition for x
		}
	}

	Jacobi_Solver(x,b);

	if(verbose){
		cout<<"\n\nx for Jacobi:\n";
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				cout<<x[I(i,j)]<<", ";
			}
			cout<<std::endl;
		}	
	}
	cout<<"\n\n";

	//////////////////////////////////////////////////////////////////////////
	////test Gauss-Seidel
	memset(x,0x0000,sizeof(double)*s);
	for(int i=-1;i<=n;i++){
		for(int j=-1;j<=n;j++){
			if(B(i,j))
				x[I(i,j)]=(double)(i*i+j*j);	////set boundary condition for x
		}
	}

	Gauss_Seidel_Solver(x,b);

	if(verbose){
		cout<<"\n\nx for Gauss-Seidel:\n";
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				cout<<x[I(i,j)]<<", ";
			}
			cout<<std::endl;
		}	
	}
	cout<<"\n\n";

	//////////////////////////////////////////////////////////////////////////
	////test Red-Black Gauss-Seidel
	memset(x,0x0000,sizeof(double)*s);
	for(int i=-1;i<=n;i++){
		for(int j=-1;j<=n;j++){
			if(B(i,j))
				x[I(i,j)]=(double)(i*i+j*j);	////set boundary condition for x
		}
	}

	Red_Black_Gauss_Seidel_Solver(x,b);

	if(verbose){
		cout<<"\n\nx for Red-Black Gauss-Seidel:\n";
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				cout<<x[I(i,j)]<<", ";
			}
			cout<<std::endl;
		}	
	}
	cout<<"\n\n";

	//////////////////////////////////////////////////////////////////////////

	delete [] x;
	delete [] b;
}

//////////////////////////////////////////////////////////////////////////
////TODO 1: your GPU variables and functions start here

////Your implementations end here
//////////////////////////////////////////////////////////////////////////

ofstream out;

//////////////////////////////////////////////////////////////////////////
////GPU test function
void Test_GPU_Solver()
{
	double* x=new double[s];
	memset(x,0x0000,sizeof(double)*s);
	double* b=new double[s];

	//////////////////////////////////////////////////////////////////////////
	////initialize x and b
	for(int i=-1;i<=n;i++){
		for(int j=-1;j<=n;j++){
			b[I(i,j)]=4.0;		////set the values for the right-hand side
		}
	}
	for(int i=-1;i<=n;i++){
		for(int j=-1;j<=n;j++){
			if(B(i,j))
				x[I(i,j)]=(double)(i*i+j*j);	////set boundary condition for x
		}
	}

	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	float gpu_time=0.0f;
	hipDeviceSynchronize();
	hipEventRecord(start);

	//////////////////////////////////////////////////////////////////////////
	////TODO 2: call your GPU functions here
	////Requirement: You need to copy data from the CPU arrays, conduct computations on the GPU, and copy the values back from GPU to CPU
	////The final positions should be stored in the same place as the CPU function, i.e., the array of x
	////The correctness of your simulation will be evaluated by the residual (<1e-3)
	//////////////////////////////////////////////////////////////////////////

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time,start,end);
	printf("\nGPU runtime: %.4f ms\n",gpu_time);
	hipEventDestroy(start);
	hipEventDestroy(end);
	//////////////////////////////////////////////////////////////////////////

	////output x
	if(verbose){
		cout<<"\n\nx for your GPU solver:\n";
		for(int i=0;i<n;i++){
			for(int j=0;j<n;j++){
				cout<<x[I(i,j)]<<", ";
			}
			cout<<std::endl;
		}	
	}

	////calculate residual
	double residual=0.0;
	for(int i=0;i<n;i++){
		for(int j=0;j<n;j++){
			residual+=pow(4.0*x[I(i,j)]-x[I(i-1,j)]-x[I(i+1,j)]-x[I(i,j-1)]-x[I(i,j+1)]-b[I(i,j)],2);
		}
	}
	cout<<"\n\nresidual for your GPU solver: "<<residual<<endl;

	out<<"R0: "<<residual<<endl;
	out<<"T1: "<<gpu_time<<endl;

	//////////////////////////////////////////////////////////////////////////

	delete [] x;
	delete [] b;
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_competition_3_linear_solver.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Test_CPU_Solvers();	////You may comment out this line to run your GPU solver only
	Test_GPU_Solver();	////Test function for your own GPU implementation

	return 0;
}